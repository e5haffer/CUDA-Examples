#include "hip/hip_runtime.h"
/*
 * Copyright 1993-2014 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

#include "defines.h"
#include "fluidsGL_kernels.h"

#include <stdio.h>
#include <stdlib.h>

#include <hipfft/hipfft.h>          // CUDA FFT Libraries
//#include <hip/hip_runtime_api.h>    // Helper functions for CUDA Error handling


// Texture reference for reading velocity field
texture<float2, 2> texref;
static hipArray *array = NULL;

void setup_texture(int x, int y)
{

    // Wrap mode appears to be the new default
    texref.filterMode = hipFilterModeLinear;
    hipChannelFormatDesc desc = hipCreateChannelDesc<float2>();

    hipMallocArray(&array, &desc, y, x);
}

void bind_texture(void)
{
    hipBindTextureToArray(texref, array);
}

void unbind_texture(void)
{
    hipUnbindTexture(texref);
}

void delete_texture(void)
{
    hipFreeArray(array);
}

void update_texture(float2 *data, size_t wib, size_t h, size_t pitch)
{
    hipMemcpy2DToArray(array, 0, 0, data, pitch, wib, h, hipMemcpyDeviceToDevice);
}

// Note that these kernels are designed to work with arbitrary
// domain sizes, not just domains that are multiples of the tile
// size. Therefore, we have extra code that checks to make sure
// a given thread location falls within the domain boundaries in
// both X and Y. Also, the domain is covered by looping over
// multiple elements in the Y direction, while there is a one-to-one
// mapping between threads in X and the tile size in X.
// Nolan Goodnight 9/22/06

// This method adds constant force vectors to the velocity field
// stored in 'v' according to v(x,t+1) = v(x,t) + dt * f.
__global__ void
addForces_k(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r, size_t pitch)
{

    int tx = threadIdx.x;
    int ty = threadIdx.y;
    float2 *fj = (float2 *)((char *)v + (ty + spy) * pitch) + tx + spx;

    float2 vterm = *fj;
    tx -= r;
    ty -= r;
    float s = 1.f / (1.f + tx*tx*tx*tx + ty*ty*ty*ty);
    vterm.x += s * fx;
    vterm.y += s * fy;
    *fj = vterm;
}

// This method performs the velocity advection step, where we
// trace velocity vectors back in time to update each grid cell.
// That is, v(x,t+1) = v(p(x,-dt),t). Here we perform bilinear
// interpolation in the velocity space.
__global__ void
advectVelocity_k(float2 *v, float *vx, float *vy,
                 int dx, int pdx, int dy, float dt, int lb)
{

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float2 vterm, ploc;
    float vxterm, vyterm;

    // gtidx is the domain location in x for this thread
    if (gtidx < dx)
    {
        for (p = 0; p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fj = fi * pdx + gtidx;
                vterm = tex2D(texref, (float)gtidx, (float)fi);
                ploc.x = (gtidx + 0.5f) - (dt * vterm.x * dx);
                ploc.y = (fi + 0.5f) - (dt * vterm.y * dy);
                vterm = tex2D(texref, ploc.x, ploc.y);
                vxterm = vterm.x;
                vyterm = vterm.y;
                vx[fj] = vxterm;
                vy[fj] = vyterm;
            }
        }
    }
}

// This method performs velocity diffusion and forces mass conservation
// in the frequency domain. The inputs 'vx' and 'vy' are complex-valued
// arrays holding the Fourier coefficients of the velocity field in
// X and Y. Diffusion in this space takes a simple form described as:
// v(k,t) = v(k,t) / (1 + visc * dt * k^2), where visc is the viscosity,
// and k is the wavenumber. The projection step forces the Fourier
// velocity vectors to be orthogonal to the vectors for each
// wavenumber: v(k,t) = v(k,t) - ((k dot v(k,t) * k) / k^2.
__global__ void
diffuseProject_k(float2 *vx, float2 *vy, int dx, int dy, float dt,
                 float visc, int lb)
{

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float2 xterm, yterm;

    // gtidx is the domain location in x for this thread
    if (gtidx < dx)
    {
        for (p = 0; p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fj = fi * dx + gtidx;
                xterm = vx[fj];
                yterm = vy[fj];

                // Compute the index of the wavenumber based on the
                // data order produced by a standard NN FFT.
                int iix = gtidx;
                int iiy = (fi>dy/2)?(fi-(dy)):fi;

                // Velocity diffusion
                float kk = (float)(iix * iix + iiy * iiy); // k^2
                float diff = 1.f / (1.f + visc * dt * kk);
                xterm.x *= diff;
                xterm.y *= diff;
                yterm.x *= diff;
                yterm.y *= diff;

                // Velocity projection
                if (kk > 0.f)
                {
                    float rkk = 1.f / kk;
                    // Real portion of velocity projection
                    float rkp = (iix * xterm.x + iiy * yterm.x);
                    // Imaginary portion of velocity projection
                    float ikp = (iix * xterm.y + iiy * yterm.y);
                    xterm.x -= rkk * rkp * iix;
                    xterm.y -= rkk * ikp * iix;
                    yterm.x -= rkk * rkp * iiy;
                    yterm.y -= rkk * ikp * iiy;
                }

                vx[fj] = xterm;
                vy[fj] = yterm;
            }
        }
    }
}

// This method updates the velocity field 'v' using the two complex
// arrays from the previous step: 'vx' and 'vy'. Here we scale the
// real components by 1/(dx*dy) to account for an unnormalized FFT.
__global__ void
updateVelocity_k(float2 *v, float *vx, float *vy,
                 int dx, int pdx, int dy, int lb, size_t pitch)
{

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    float vxterm, vyterm;
    float2 nvterm;

    // gtidx is the domain location in x for this thread
    if (gtidx < dx)
    {
        for (p = 0; p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fjr = fi * pdx + gtidx;
                vxterm = vx[fjr];
                vyterm = vy[fjr];

                // Normalize the result of the inverse FFT
                float scale = 1.f / (dx * dy);
                nvterm.x = vxterm * scale;
                nvterm.y = vyterm * scale;

                float2 *fj = (float2 *)((char *)v + fi * pitch) + gtidx;
                *fj = nvterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}

// This method updates the particles by moving particle positions
// according to the velocity field and time step. That is, for each
// particle: p(t+1) = p(t) + dt * v(p(t)).
__global__ void
advectParticles_k(float2 *part, float2 *v, int dx, int dy,
                  float dt, int lb, size_t pitch)
{

    int gtidx = blockIdx.x * blockDim.x + threadIdx.x;
    int gtidy = blockIdx.y * (lb * blockDim.y) + threadIdx.y * lb;
    int p;

    // gtidx is the domain location in x for this thread
    float2 pterm, vterm;

    if (gtidx < dx)
    {
        for (p = 0; p < lb; p++)
        {
            // fi is the domain location in y for this thread
            int fi = gtidy + p;

            if (fi < dy)
            {
                int fj = fi * dx + gtidx;
                pterm = part[fj];

                int xvi = ((int)(pterm.x * dx));
                int yvi = ((int)(pterm.y * dy));
                vterm = *((float2 *)((char *)v + yvi * pitch) + xvi);

                pterm.x += dt * vterm.x;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.x += 1.f;
                pterm.x = pterm.x - (int)pterm.x;
                pterm.y += dt * vterm.y;
                pterm.y = pterm.y - (int)pterm.y;
                pterm.y += 1.f;
                pterm.y = pterm.y - (int)pterm.y;

                part[fj] = pterm;
            }
        } // If this thread is inside the domain in Y
    } // If this thread is inside the domain in X
}


void addForces(float2 *v, int dx, int dy, int spx, int spy, float fx, float fy, int r)
{

    dim3 tids(2*r+1, 2*r+1);

    addForces_k<<<1, tids>>>(v, dx, dy, spx, spy, fx, fy, r, tPitch);
}


void advectVelocity(float2 *v, float *vx, float *vy, int dx, int pdx, int dy, float dt)
{
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));

    dim3 tids(TIDSX, TIDSY);

    update_texture(v, DIM*sizeof(float2), DIM, tPitch);
    advectVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, dt, TILEY/TIDSY);

}


void diffuseProject(float2 *vx, float2 *vy, int dx, int dy, float dt, float visc,
                    hipfftHandle planr2c, hipfftHandle planc2r)
{
    // Forward FFT
    hipfftExecR2C(planr2c, (hipfftReal *)vx, (hipfftComplex *)vx);
    hipfftExecR2C(planr2c, (hipfftReal *)vy, (hipfftComplex *)vy);

    uint3 grid = make_uint3((dx/TILEX)+(!(dx%TILEX)?0:1),
                            (dy/TILEY)+(!(dy%TILEY)?0:1), 1);
    uint3 tids = make_uint3(TIDSX, TIDSY, 1);

    diffuseProject_k<<<grid, tids>>>(vx, vy, dx, dy, dt, visc, TILEY/TIDSY);

    // Inverse FFT
    hipfftExecC2R(planc2r, (hipfftComplex *)vx, (hipfftReal *)vx);
    hipfftExecC2R(planc2r, (hipfftComplex *)vy, (hipfftReal *)vy);
}


void updateVelocity(float2 *v, float *vx, float *vy, int dx, int pdx, int dy)
{
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));
    dim3 tids(TIDSX, TIDSY);

    updateVelocity_k<<<grid, tids>>>(v, vx, vy, dx, pdx, dy, TILEY/TIDSY, tPitch);
}


void advectParticles(struct hipGraphicsResource *cuda_vbo_resource, float2 *v, int dx, int dy, float dt)
{
    dim3 grid((dx/TILEX)+(!(dx%TILEX)?0:1), (dy/TILEY)+(!(dy%TILEY)?0:1));
    dim3 tids(TIDSX, TIDSY);

    float2 *p;
    hipGraphicsMapResources(1, &cuda_vbo_resource, 0);

    size_t num_bytes;
    hipGraphicsResourceGetMappedPointer((void **)&p, &num_bytes,
                                         cuda_vbo_resource);

    advectParticles_k<<<grid, tids>>>(p, v, dx, dy, dt, TILEY/TIDSY, tPitch);

    hipGraphicsUnmapResources(1, &cuda_vbo_resource, 0);
}
